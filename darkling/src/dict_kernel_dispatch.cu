#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "darkling_rules.h"
#include "darkling_device_queue.h"
#include "darkling_telemetry.h"
#include "hash_primitives.cuh"
#include "kernel_optimizations.cuh"

#ifdef DARK_ENABLE_PTX_RULES
// PTX rule function declarations
extern "C" __device__ void rule_prefix_1_ptx(uint8_t*, const uint8_t*, uint32_t, const RuleParams*, uint32_t, uint32_t);
extern "C" __device__ void rule_suffix_d4_ptx(uint8_t*, const uint8_t*, uint32_t, const RuleParams*, uint32_t, uint32_t);
extern "C" __device__ void rule_vectorized_copy_ptx(uint8_t*, const uint8_t*, uint32_t, const RuleParams*, uint32_t, uint32_t);
extern "C" __device__ void rule_case_toggle_simd_ptx(uint8_t*, const uint8_t*, uint32_t, const RuleParams*, uint32_t, uint32_t);
extern "C" __device__ void rule_leet_lookup_optimized_ptx(uint8_t*, const uint8_t*, uint32_t, const RuleParams*, uint32_t, uint32_t);
#endif

extern "C" __device__ uint32_t rule_prefix_1(uint8_t* dst, const uint8_t* src, uint32_t len,
                                             const RuleParams* params, uint32_t variant_idx, uint32_t variant_count);
extern "C" __device__ uint32_t rule_suffix_d4(uint8_t* dst, const uint8_t* src, uint32_t len,
                                               const RuleParams* params, uint32_t variant_idx, uint32_t variant_count);

__global__ void persistent_kernel(const uint8_t* words, const uint32_t* offsets, DlTelemetry* tel) {
  DlWorkItem item;
  uint32_t max_queue_size = 0;
  while (dq_pop(&item)) {
    uint32_t current_size = dq_size();
    if (current_size > max_queue_size) {
      max_queue_size = current_size;
      atomicMax((unsigned long long*)&tel->queue_max_size, (unsigned long long)current_size);
    }
    if (item.word_count == 0) continue;
    uint32_t end = offsets[item.word_start + item.word_count];
    uint32_t last = offsets[item.word_start + item.word_count - 1];
    if (end <= last) {
      printf("missing sentinel for word range %u-%u\n", item.word_start, item.word_start + item.word_count);
      continue;
    }
    for (uint32_t i = 0; i < item.word_count; ++i) {
      uint32_t idx = item.word_start + i;
      const uint8_t* src = words + offsets[idx];
      uint32_t len = offsets[idx+1] - offsets[idx] - 1; // strip \n
      DlRuleMC rule = g_rules[item.rule_start];
      RuleParams params;
      #pragma unroll
      for (int j = 0; j < 16; ++j) params.bytes[j] = rule.params[j];
      DlRuleDispatch disp = g_dispatch[rule.shape];
      uint8_t tmp[128];
      for (uint32_t v = 0; v < 1; ++v) {
        uint32_t candidate_len = disp.fn(tmp, src, len, &params, v, disp.variants);
        if (candidate_len > 120) continue;
        uint32_t dig[4];
        md5_hash(tmp, candidate_len, dig);
        atomicAdd(&tel->candidates_generated, 1ULL);
        if (check_hash(dig)) {
          atomicAdd(&tel->hits, 1ULL);
        }
      }
      atomicAdd(&tel->words_processed, 1ULL);
    }
  }
}

void launch_persistent(const uint8_t* d_words, const uint32_t* d_offsets, DlTelemetry* tel) {
  int device;
  hipGetDevice(&device);
  
  // Use optimized launch parameters
  KernelOptParams opt_params = get_optimal_launch_params(device);
  
  // Configure shared memory for optimal performance
  hipFuncSetAttribute(reinterpret_cast<const void*>(persistent_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, opt_params.shared_mem_bytes);
  
  // Launch with optimized parameters
  persistent_kernel<<<opt_params.blocks, opt_params.threads_per_block, opt_params.shared_mem_bytes>>>(d_words, d_offsets, tel);
}
