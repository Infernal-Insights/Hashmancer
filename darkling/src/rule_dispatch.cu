#include <hip/hip_runtime.h>
#include "darkling_rules.h"

extern "C" __device__ uint32_t rule_prefix_1(uint8_t* dst, const uint8_t* src, uint32_t len,
                                             const RuleParams* params, uint32_t variant_idx, uint32_t variant_count) {
  if (len > 126) return len;
  static __constant__ uint8_t table[14] = { '!', '@', '#', '$', '%', '^', '&', '*', '(', ')', '_', '+', '=', '-' };
  uint8_t sym = table[params->bytes[0] % 14];
  dst[0] = sym;
  for (uint32_t i = 0; i < len; ++i) dst[i+1] = src[i];
  return len + 1;
}

extern "C" __device__ uint32_t rule_suffix_d4(uint8_t* dst, const uint8_t* src, uint32_t len,
                                              const RuleParams* params, uint32_t variant_idx, uint32_t variant_count) {
  if (len > 123) return len;
  for (uint32_t i = 0; i < len; ++i) dst[i] = src[i];
  uint32_t v = variant_idx;
  dst[len+0] = '0' + (v % 10);
  dst[len+1] = '0' + ((v/10) % 10);
  dst[len+2] = '0' + ((v/100) % 10);
  dst[len+3] = '0' + ((v/1000) % 10);
  return len + 4;
}
