#include <assert.h>
#include <hip/hip_runtime.h>
#include "darkling_rules.h"
#include "darkling_device_queue.h"
#include "darkling_telemetry.h"

void launch_persistent(const uint8_t*, const uint32_t*, DlTelemetry*);

int main() {
  dl_rules_load_json("configs/ruleset_baked256.json");
  const char* word = "test\n";
  uint32_t offsets[2] = {0,5};
  uint8_t* d_words; uint32_t* d_offsets; DlTelemetry* tel;
  hipMalloc(&d_words,5); hipMemcpy(d_words,word,5,hipMemcpyHostToDevice);
  hipMalloc(&d_offsets,sizeof(offsets)); hipMemcpy(d_offsets,offsets,sizeof(offsets),hipMemcpyHostToDevice);
  hipMalloc(&tel,sizeof(DlTelemetry)); hipMemset(tel,0,sizeof(DlTelemetry));
  DlWorkItem item{0,1,0,1};
  uint32_t zero=0, tail=1;
  hipMemcpyToSymbol(HIP_SYMBOL(g_qhead),&zero,sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(g_qtail),&zero,sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(g_queue),&item,sizeof(DlWorkItem));
  hipMemcpyToSymbol(HIP_SYMBOL(g_qtail),&tail,sizeof(uint32_t));
  launch_persistent(d_words,d_offsets,tel);
  hipDeviceSynchronize();
  DlTelemetry h; hipMemcpy(&h,tel,sizeof(h),hipMemcpyDeviceToHost);
  assert(h.words_processed==1);
  hipFree(tel); hipFree(d_words); hipFree(d_offsets);
  return 0;
}
